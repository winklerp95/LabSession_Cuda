#include "hip/hip_runtime.h"
/**
 * Matrix multiplication: C = A * B.
 *     where A, B and C are NxN matrices
 *
 * This sample is a very basic sample that implements a matrix multiplication
 */

#include <stdio.h>
#include <chrono>
#include <iostream>
#include <string>
#include <cstdlib>

 // For the CUDA runtime routines (prefixed with "cuda_")
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

// Own headers
#include "./cudaKernel.cuh"


using namespace std;


__global__
void matrixMulKernel(const float* A, const float* B, float* C,
    int numElements) {

    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;

    float tmpSum = 0;

    if (ROW < numElements && COL < numElements) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < numElements; i++) {
            tmpSum += A[ROW * numElements + i] * B[i * numElements + COL];
        }

        C[ROW * numElements + COL] = tmpSum;
    }
    
}


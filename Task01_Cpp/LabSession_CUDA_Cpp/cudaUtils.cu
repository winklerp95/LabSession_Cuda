#include <stdio.h>
#include <iostream>
#include <vector>
#include <sstream>

// For the CUDA runtime routines (prefixed with "cuda_")
#include "hip/hip_runtime.h"
#include "./cudaUtils.cuh"

using namespace std;

void checkError(hipError_t err, const char* operation) {
    if (err != hipSuccess) {
        std::cerr << "Error during " << operation << ": " << err << std::endl;
        std::cerr << hipGetErrorString(err);
        exit(err);
    }
}

vector<string> runCudaGetDevices() {

    vector<string> deviceInfoVector;

    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        cout << "No CUDA-capable devices found";
    }
    else {
        cout << "Devices found: " << endl;

        for (int deviceID = 0; deviceID < deviceCount; ++deviceID) {
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, deviceID);

            deviceInfoVector.push_back(deviceProp.name);
            cout << "  " << deviceProp.name << endl;
        }
    }

    return deviceInfoVector;
}